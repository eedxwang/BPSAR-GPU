#include "hip/hip_runtime.h"

#pragma comment(lib, "ws2_32.lib")

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <fstream>
#include <string>
#include <WinSock2.h>
#include <Windows.h>
#include "hipfft/hipfft.h"
#include "hipfft/hipfftw.h"
#include <hip/hip_math_constants.h>

#define pi 3.141592654

using namespace std;
typedef struct Es_Data
{
	hipfftDoubleComplex *data;
	float *phi;
	float *thi;
	double *RadarPos;
	double *XGeo;
	double *YGeo;
	int nSampling_f, nSampling_phi;
	int GeoNumberX, GeoNumberY;
	double B; 
	double f0; //for ES simulation results, this will be the first frequecny appealled.
	double R0; 
	int fft_times;
}EsDataStr;
EsDataStr Es;

typedef struct BFAInputPars
{
	int nSampling_f;
	int geoPulse;
	double maxRange;
	double R0;
	double f0;
	int times;
	int GeoNumberX, GeoNumberY;
}BFAInputParsStr;

#pragma region Pre Defination
void ReadEsFile(string filepath, int polarInc);
__global__ void BFAProcess(double2 *, double *, double *, double *, hipfftDoubleComplex *, BFAInputParsStr, int, int);
int ReadInputFile(void);
void OutputData(char *filepath, double2 *data, int len);
#pragma endregion


#pragma region /* GPU Global Variables */
__device__ hipfftDoubleComplex *gpuRangeSignal;
__device__ hipfftDoubleComplex *gpuEs;
__device__ double *gpuGeoX;
__device__ double *gpuGeoY;
__device__ double *gpuRadarPos;
__device__ double2 *gpuGeo;
#pragma endregion


int main(int argc, char *argv[])
{
	/* Read the input file */
	printf("BFA start!\r\n");
	ReadInputFile();

	/* Ask memory from GPU */
	hipMalloc((void **)&gpuRangeSignal, sizeof(hipfftDoubleComplex)*Es.nSampling_f * Es.nSampling_phi * Es.fft_times);
	hipMalloc((void **)&gpuEs, sizeof(hipfftDoubleComplex)*Es.nSampling_f * Es.nSampling_phi * Es.fft_times);
	hipMalloc((void **)&gpuGeoX, sizeof(double)* Es.GeoNumberX);
	hipMalloc((void **)&gpuGeoY, sizeof(double)* Es.GeoNumberY);
	hipMalloc((void **)&gpuRadarPos, sizeof(double)*Es.nSampling_phi * 3);
	/* Copy the Es data, X Geo and Y Geo data to the GPU RAM */
	hipMemcpy(gpuEs, Es.data, sizeof(hipfftDoubleComplex)*Es.nSampling_f * Es.nSampling_phi *  Es.fft_times, hipMemcpyHostToDevice);
	hipMemcpy(gpuGeoX, Es.XGeo, sizeof(double)* Es.GeoNumberX, hipMemcpyHostToDevice);
	hipMemcpy(gpuGeoY, Es.YGeo, sizeof(double)* Es.GeoNumberY, hipMemcpyHostToDevice);
	hipMemcpy(gpuRadarPos, Es.RadarPos, sizeof(double)*Es.nSampling_phi * 3, hipMemcpyHostToDevice); 

	/* calulate the range signal */
	printf("BFA compress signal in every pulse!");
	for (int ii = 0; ii < Es.nSampling_phi; ii++)
	{
		hipfftHandle plan;
		hipfftResult res = hipfftPlan1d(&plan, Es.nSampling_f * Es.fft_times, HIPFFT_Z2Z, 1);
		res = hipfftExecZ2Z(plan, &gpuEs[ii * Es.nSampling_f * Es.fft_times], &gpuRangeSignal[ii * Es.nSampling_f * Es.fft_times], HIPFFT_FORWARD);
		hipDeviceSynchronize();
		hipfftDestroy(plan);
	}
	/* fftshift and conj*/
	hipMemcpy(Es.data, gpuRangeSignal, sizeof(hipfftDoubleComplex)*Es.nSampling_f * Es.nSampling_phi *  Es.fft_times, hipMemcpyDeviceToHost);
	for (int jj = 0; jj < Es.nSampling_phi; jj++)
	{
		long phiInc = jj * Es.nSampling_f * Es.fft_times;
		for (int ii = 0; ii < Es.nSampling_f * Es.fft_times / 2; ii++)
		{
			hipfftDoubleComplex tmp; tmp.x = Es.data[ii + phiInc].x; tmp.y = Es.data[ii + phiInc].y;
			Es.data[ii + phiInc].x = Es.data[ii + phiInc + Es.nSampling_f * Es.fft_times / 2].x;
			Es.data[ii + phiInc].y = Es.data[ii + phiInc + Es.nSampling_f * Es.fft_times / 2].y * (-1);
			Es.data[ii + phiInc + Es.nSampling_f * Es.fft_times / 2].x = tmp.x ;
			Es.data[ii + phiInc + Es.nSampling_f * Es.fft_times / 2].y = tmp.y * (-1);
		}
	}
	/*Copy to GPU RAM*/
	hipMemcpy(gpuRangeSignal, Es.data, sizeof(hipfftDoubleComplex)*Es.nSampling_f * Es.nSampling_phi *  Es.fft_times, hipMemcpyHostToDevice);
	printf("  done! \r\n");

	/*start BFA*/
	/*Setup GPU RAM for saving the imaging result*/
	printf("BFA start bp algorithm!");
	 hipMalloc((void **)&gpuGeo, sizeof(double2)*Es.GeoNumberX*Es.GeoNumberY);
	 /*Write a Struct to transmit the setup parameters to the CUDA function*/
	 BFAInputParsStr parStr; // set the input parameter for CUDA.
	 parStr.f0 = Es.f0; parStr.geoPulse = Es.nSampling_phi; parStr.maxRange = 0.3 / Es.B*Es.nSampling_f / 4;
	 parStr.nSampling_f = Es.nSampling_f; parStr.R0 = Es.R0; parStr.times = Es.fft_times;
	 parStr.GeoNumberX = Es.GeoNumberX; parStr.GeoNumberY = Es.GeoNumberY;

	 /* If the pixel needed is too large, we need to cut the batch
	 The max number of X and Y is 500 and 500. 	 */
	 /*Determinate how many batches are needed*/
	int XBatch = (int)((double)Es.GeoNumberX / 500.0);
	int YBatch = (int)((double)Es.GeoNumberY / 500.0);
	/*Less than mini possible range, do not need batch*/
	if (XBatch == 0 && YBatch == 0)
	{
		/*Run BFA Cuda Funtion*/
		dim3 threadPerBlock(Es.GeoNumberY, 1, 1);
		dim3 numBlock(Es.GeoNumberX, 1, 1);
		BFAProcess << <numBlock, threadPerBlock>> >(gpuGeo, gpuRadarPos,
			gpuGeoX, gpuGeoY, gpuRangeSignal, parStr, 0, 0);
		hipError_t err1 = hipPeekAtLastError();
		hipError_t err2 = hipDeviceSynchronize();
	}
	else
	{
		/*larger than max range*/
		for (int jj = 0; jj < (YBatch); jj++)
		{
			for (int ii = 0; ii < (XBatch); ii++)
			{
				dim3 threadPerBlock(500, 1, 1);
				dim3 numBlock(500, 1, 1);
				BFAProcess << <numBlock, threadPerBlock >> >(gpuGeo, gpuRadarPos,
							gpuGeoX, gpuGeoY, gpuRangeSignal, parStr, 500*ii, 500*jj);
				hipError_t err1 = hipPeekAtLastError();
				hipError_t err2 = hipDeviceSynchronize();
			}
		}
		/*Process margin batch*/
		/* X range */
		for (int ii = 0; ii < (XBatch); ii++)
		{
			dim3 threadPerBlock(Es.GeoNumberY - 500 * (YBatch), 1, 1);
			dim3 numBlock(500, 1, 1);
			BFAProcess << <numBlock, threadPerBlock >> >(gpuGeo, gpuRadarPos,
				gpuGeoX, gpuGeoY, gpuRangeSignal, parStr, 500 * ii, 500 * (YBatch));
			hipError_t err1 = hipPeekAtLastError();
			hipError_t err2 = hipDeviceSynchronize();
		}
		/* Y range */
		for (int ii = 0; ii < (YBatch); ii++)
		{
			dim3 threadPerBlock(500, 1, 1);
			dim3 numBlock(Es.GeoNumberX - 500 * (XBatch), 1, 1);
			BFAProcess << <numBlock, threadPerBlock >> >(gpuGeo, gpuRadarPos,
				gpuGeoX, gpuGeoY, gpuRangeSignal, parStr, 500 * (XBatch), 500 * ii);
			hipError_t err1 = hipPeekAtLastError();
			hipError_t err2 = hipDeviceSynchronize();
		}
		/*In the corner*/
		dim3 threadPerBlock(Es.GeoNumberY - 500 * (YBatch), 1, 1);
		dim3 numBlock(Es.GeoNumberX - 500 * (XBatch), 1, 1);
		BFAProcess << <numBlock, threadPerBlock >> >(gpuGeo, gpuRadarPos,
			gpuGeoX, gpuGeoY, gpuRangeSignal, parStr, 500 * (XBatch), 500 * (YBatch));
		hipError_t err1 = hipPeekAtLastError();
		hipError_t err2 = hipDeviceSynchronize();
	}
	/*display result*/
	/*Setup CPU RAM to save the imaging result*/
	double2 *resultBFA2 = (double2 *)malloc(sizeof(double2)*Es.GeoNumberX*Es.GeoNumberY);
	hipError_t err3 = 
	hipMemcpy(resultBFA2, gpuGeo, sizeof(double2)*Es.GeoNumberX*Es.GeoNumberY, hipMemcpyDeviceToHost);
	printf("  done! \r\n");
	printf("BFA output data!");
	OutputData("output.txt", resultBFA2, Es.GeoNumberX*Es.GeoNumberY);
	printf("  done! \r\n");
	/*Not all the RAM are free, but, since this function will end, there is no need to free it.
	* Please note that, all GPU RAM are free.
	*/
	hipFree(gpuRangeSignal);
	hipFree(gpuEs);
	hipFree(gpuGeoX);
	hipFree(gpuGeoY);
	hipFree(gpuRadarPos);
	hipFree(gpuGeo);

	return 0;
}

__global__ void BFAProcess(double2 *geo, double *pos,
										double *xgeo, double *ygeo,
										hipfftDoubleComplex *range, BFAInputParsStr parStr, int xBatch, int yBatch)
{
	int xi = blockIdx.x;
	int patchi = threadIdx.x;

	double RadarPostionParameterX, RadarPostionParameterY, RadarPostionParameterZ;
	double distance;
	long position;

	geo[(xi + xBatch) + (patchi + yBatch)*parStr.GeoNumberX].x = 0;
	geo[(xi + xBatch) + (patchi + yBatch)*parStr.GeoNumberX].y = 0;

	/*Calculate every pulse*/
	for (int pulseNum = 0; pulseNum < parStr.geoPulse; pulseNum++)
	{
		/*Calculate the distance between the current radar pos and 
		current pixel of SAR image*/
		distance = 0;
		RadarPostionParameterX = pos[pulseNum * 3 + 0];
		RadarPostionParameterY = pos[pulseNum * 3 + 1];
		RadarPostionParameterZ = pos[pulseNum * 3 + 2];
		distance = (RadarPostionParameterX - xgeo[xi + xBatch])*(RadarPostionParameterX - xgeo[xi + xBatch]);
		distance = (RadarPostionParameterY - ygeo[patchi + yBatch]) * (RadarPostionParameterY - ygeo[patchi + yBatch]) + distance;
		distance = RadarPostionParameterZ * RadarPostionParameterZ + distance;
		distance = sqrt(distance);
		/*Deteminate where is the range*/
		position = (long)floor(((distance - parStr.R0) / (parStr.maxRange * 2) + 1.0 / 2.0)*parStr.nSampling_f * parStr.times);
		/*Coherent Accumulation*/
		geo[(xi + xBatch) + (patchi + yBatch)*parStr.GeoNumberX].x = range[position + pulseNum*parStr.nSampling_f * parStr.times].x * cos(4 * pi * parStr.f0 / 0.3*(distance))
			- range[position + pulseNum*parStr.nSampling_f * parStr.times].y*sin(4 * pi * parStr.f0 / 0.3*(distance)) + geo[(xi + xBatch) + (patchi + yBatch)*parStr.GeoNumberX].x;
		geo[(xi + xBatch) + (patchi + yBatch)*parStr.GeoNumberX].y = range[position + pulseNum*parStr.nSampling_f * parStr.times].x * sin(4 * pi * parStr.f0 / 0.3*(distance))
			+ range[position + pulseNum*parStr.nSampling_f *parStr.times].y*cos(4 * pi * parStr.f0 / 0.3*(distance)) + geo[(xi + xBatch) + (patchi + yBatch)*parStr.GeoNumberX].y;
	}
}

void OutputData(char *filepath, double2 *data, int len)
{
	ofstream OutFile;
	OutFile.open(filepath);

	char *outputBuf = (char *)malloc(20 * len * 2);

	long pos = 0; int num = 0;
	num = sprintf(&outputBuf[pos], " %d %d\r\n", Es.GeoNumberX, Es.GeoNumberY);
	pos = pos + num;
	for (int ii = 0; ii < len; ii++)
	{
		num = sprintf(&outputBuf[pos], " %g %g\r\n", data[ii].x, data[ii].y);
		pos = pos + num;
		//OutFile << data[ii].x << ' ' << data[ii].y << endl;
	}
	OutFile.write(outputBuf, pos);
	OutFile.close();
}

void ReadEsFile(string filepath, int polarInc)
{
	ifstream EsFile;
	char output[200];
	char number[15][30];

	Es.data = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex)*Es.nSampling_f*Es.nSampling_phi*Es.fft_times);
	Es.phi = (float *)malloc(sizeof(float)*Es.nSampling_f*Es.nSampling_phi);
	Es.thi = (float *)malloc(sizeof(float)*Es.nSampling_f*Es.nSampling_phi);

	//zeros all memory
	for (int ii = 0; ii < Es.nSampling_f * Es.nSampling_phi * Es.fft_times; ii++)
	{
		Es.data[ii].x = 0; 	Es.data[ii].y = 0;
	}

	EsFile.open(filepath);
	EsFile.getline(output, 200);
	for (int i = 0; i < Es.nSampling_phi; i++)
	{
		for (int j = 0; j < Es.nSampling_f; j++)
		{
			EsFile >> number[0] >> number[1] >> number[2] >> number[3] >> number[4] >> number[5]
				>> number[6] >> number[7] >> number[8] >> number[9] >> number[10] >> number[11]
				>> number[12] >> number[13] >> number[14];
			Es.data[j + i*Es.nSampling_f * Es.fft_times].x = atof(number[polarInc - 1]);
			Es.data[j + i*Es.nSampling_f * Es.fft_times].y = atof(number[polarInc]);
			Es.thi[j + i*Es.nSampling_f] = atof(number[1]) / 180 * pi;
			Es.phi[j + i*Es.nSampling_f] = atof(number[2]) / 180 * pi;
			// get the center frequency.
			if (i == 0 && j == 0)	Es.f0 = atof(number[0]);
			//get the bandwidth.
			if (i == 0 && j == (Es.nSampling_f - 1)) Es.B = atof(number[0]) - Es.f0;
		}
	}
	EsFile.close();
}

int ReadInputFile(void)
{
	ifstream InputFile; 	
	char output[200]; //characters buffer
	char number[6][30]; //number buffer

	InputFile.open("bfa_input.txt");
	printf("BFA read input file!\r\n");

	if (InputFile.fail())
	{
		printf("Cannot open necessary setup file: bfa_input.txt. \r\n");
		return 0;
	}
	// ignore first 9 lines.
	for (int ii = 0; ii < 9; ii ++)
		InputFile.getline(output, 200);

	//##1 get file path
	string esFilePath;
	InputFile >> esFilePath;

	InputFile.getline(output, 200); InputFile.getline(output, 200); InputFile.getline(output, 200); //ignore 3 lines.
	//##2 read X range
	InputFile >> number[0] >> number[1] >> number[2];
	Es.GeoNumberX = floor((atof(number[1]) - atof(number[0]))/atof(number[2]));
	Es.XGeo = (double *)malloc(sizeof(double)*Es.GeoNumberX);
	for (int ii = 0; ii < Es.GeoNumberX; ii++)
		Es.XGeo[ii] = atof(number[0]) + atof(number[2])*ii;

	InputFile.getline(output, 200); InputFile.getline(output, 200); InputFile.getline(output, 200); //ignore 3 lines.
	//##3 read Y range
	InputFile >> number[0] >> number[1] >> number[2];
	Es.GeoNumberY = floor((atof(number[1]) - atof(number[0])) / atof(number[2]));
	Es.YGeo = (double *)malloc(sizeof(double)*Es.GeoNumberY);
	for (int ii = 0; ii < Es.GeoNumberY; ii++)
		Es.YGeo[ii] = atof(number[0]) + atof(number[2])*ii;

	InputFile.getline(output, 200); InputFile.getline(output, 200); InputFile.getline(output, 200); //ignore 3 lines.
	//##4 read size of Es file.
	InputFile >> number[0] >> number[1];
	Es.nSampling_f = (int)(atof(number[0]));
	Es.nSampling_phi = (int)(atof(number[1]));
	
	InputFile.getline(output, 200); InputFile.getline(output, 200); InputFile.getline(output, 200); //ignore 3 lines.
	//##5 read polar
	InputFile.getline(output, 200);
	int polarInc = 0;
	if (output[0] == 'V' && output[1] == 'V') polarInc = 5;
	if (output[0] == 'H' && output[1] == 'H') polarInc = 8;
	if (output[0] == 'V' && output[1] == 'H') polarInc = 11;
	if (output[0] == 'H' && output[1] == 'V') polarInc = 14;

	InputFile.getline(output, 200); InputFile.getline(output, 200); //ignore 3 lines.
	//##6 Set the distance between the APC and scene center
	InputFile >> number[0];
	Es.R0 = atof(number[0]);

	InputFile.getline(output, 200); InputFile.getline(output, 200); InputFile.getline(output, 200); //ignore 3 lines.
	//##7 Set the points of fouries for range compression. 
	InputFile >> number[0];
	Es.fft_times = (int)(atof(number[0]));

	//Read Es File
	printf("BFA reading es file!");
	ReadEsFile(esFilePath, polarInc);
	printf("  done!\r\n");

	//Calu the Radar Pos
	printf("BFA calu radar position for every pulse!");
	Es.RadarPos = (double *)malloc(sizeof(double)*Es.nSampling_phi * 3);
	for (int ii = 0; ii < Es.nSampling_phi; ii++)
	{	
		Es.RadarPos[ii * 3 + 0] = Es.R0 * sin(Es.phi[Es.nSampling_f * ii]) * sin(Es.thi[Es.nSampling_f * ii]);
		Es.RadarPos[ii * 3 + 1] = Es.R0 * cos(Es.phi[Es.nSampling_f * ii]) * sin(Es.thi[Es.nSampling_f * ii]);
		Es.RadarPos[ii * 3 + 2] = Es.R0 * cos(Es.thi[Es.nSampling_f * ii]);
	}
	printf("  done!\r\n");

	return 1;
}